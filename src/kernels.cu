#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <hip/hip_fp16.h>
#include <mma.h>
//#include "libs/CudaTensorLibrary/tensor.cu"

namespace Kernels
{
    __device__ constexpr bool GUESS_HANDLES{false};

    __device__ constexpr int CHANNEL_COUNT{4};
    __device__ constexpr int CONSTANTS_COUNT{11};
    __device__ constexpr int VIEW_COUNT{8};
    __device__ constexpr int VIEW_PORTIONS{2};
    __device__ constexpr int VIEW_TOTAL_COUNT{VIEW_PORTIONS*VIEW_COUNT};
    __constant__ int constants[CONSTANTS_COUNT];
    __device__ int2 imgRes(){return {constants[0], constants[1]};}
    __device__ int2 colsRows(){return{constants[2], constants[3]};}
    __device__ int2 weightsRes(){return {constants[5], VIEW_TOTAL_COUNT};}
    __device__ int weightsSize(){return constants[6];}
    __device__ int gridSize(){return constants[5];}
    __device__ int focus(){return constants[7];}
    __device__ int focusRange(){return constants[8];}
    __device__ int2 blockRadius(){return {constants[9], constants[10]};}

    __device__ constexpr int MAX_IMAGES{256};
    __device__ constexpr int MAX_SURFACES{256};
    __device__ constexpr int MAP_COUNT{2};
    __constant__ int2 focusedOffsets[MAX_IMAGES];
    __constant__ float2 offsets[MAX_IMAGES];
    __constant__ hipSurfaceObject_t inputSurfaces[MAX_SURFACES];
    __constant__ hipSurfaceObject_t outputSurfaces[VIEW_TOTAL_COUNT];
    __constant__ hipSurfaceObject_t mapSurfaces[MAP_COUNT];
    __device__ constexpr int FOCUS_MAP_IDS_COUNT{32};
    __constant__ int focusMapIDs[FOCUS_MAP_IDS_COUNT];
 
   __device__ int2 focusCoords(int2 coords, int imageID)
    {
        auto offset = focusedOffsets[imageID];
        return {coords.x+offset.x, coords.y+offset.y};
    }
    
    __device__ int2 focusCoords(int2 coords, int imageID, int focus)
    {
        auto offset = offsets[imageID];
        return {static_cast<int>(coords.x+focus*offset.x), static_cast<int>(coords.y+focus*offset.y)};
    }

    extern __shared__ half localMemory[];

    template <typename T>
    class MemoryPartitioner
    {
        public:
        __device__ MemoryPartitioner(T *inMemory)
        {
            memory = inMemory; 
        }

        __device__ T* array(int size)
        {
            T *arr = memory+consumed;
            consumed += size;
            return {arr};
        }
        private:
        T *memory;
        unsigned int consumed{0};
    };

     template <typename T>
        class PixelArray
        {
            public:
            __device__ PixelArray(){};
            __device__ PixelArray(uchar4 pixel) : channels{T(pixel.x), T(pixel.y), T(pixel.z), T(pixel.w)}{};
            __device__ PixelArray(float4 pixel) : channels{pixel.x, pixel.y, pixel.z}{};
            T channels[CHANNEL_COUNT]{0,0,0,0};
            __device__ T& operator[](int index){return channels[index];}
          
             __device__ uchar4 uch4() 
            {
                uchar4 result;
                auto data = reinterpret_cast<unsigned char*>(&result);
                for(int i=0; i<CHANNEL_COUNT; i++)
                    data[i] = __half2int_rn(channels[i]);
                return result;
            }
           
            __device__ void addWeighted(T weight, PixelArray<T> value) 
            {    
                for(int j=0; j<CHANNEL_COUNT; j++)
                    //channels[j] += value[j]*weight;
                    channels[j] = __fmaf_rn(value[j], weight, channels[j]);
            }
            
            __device__ PixelArray<T> operator/= (const T &divisor)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] /= divisor;
                return *this;
            }
            __device__ PixelArray operator-(const PixelArray &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] -= value.channels[j];
                return *this;
            }
__device__ PixelArray operator/(const float &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] /= value;
                return *this;
            }
  __device__ PixelArray operator+= (const PixelArray &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] += value.channels[j];
                return *this;
            }
        };
 
    __device__ int linearCoords(int2 coords, int width)
    {
        return coords.y*width + coords.x;
    }

    template <typename T>
    __device__ static void loadWeightsSync(T *inData, T *data)
    {
        int threadsCount = blockDim.x*blockDim.y;
        int batchSize = weightsSize()/threadsCount/2;
        int id = batchSize*linearCoords(int2{static_cast<int>(threadIdx.x), static_cast<int>(threadIdx.y)}, blockDim.x);
        if(id < weightsSize()/2)
            for(int batch=0; batch<batchSize; batch++)
            {
                int *intLocal = reinterpret_cast<int*>(data);
                int *intIn = reinterpret_cast<int*>(inData);
                intLocal[id+batch] = intIn[id+batch]; 
            }
        __syncthreads();
    }

    __device__ bool coordsOutside(int2 coords)
    {
        int2 resolution = imgRes();
        return (coords.x >= resolution.x || coords.y >= resolution.y);
    }

    __device__ int2 getImgCoords()
    {
        int2 coords;
        coords.x = (threadIdx.x + blockIdx.x * blockDim.x);
        coords.y = (threadIdx.y + blockIdx.y * blockDim.y);
        return coords;
    }
   
    template <typename T>
    __device__ PixelArray<T> loadPx(int imageID, int2 coords)
    {
        constexpr int MULT_FOUR_SHIFT{2};
        if constexpr (GUESS_HANDLES)
            return PixelArray<T>{surf2Dread<uchar4>(imageID+1, coords.x<<MULT_FOUR_SHIFT, coords.y, hipBoundaryModeClamp)};
        else    
            return PixelArray<T>{surf2Dread<uchar4>(inputSurfaces[imageID], coords.x<<MULT_FOUR_SHIFT, coords.y, hipBoundaryModeClamp)};
    }
    
    __device__ uchar4 loadPx(int imageID, int2 coords)
    {
        constexpr int MULT_FOUR_SHIFT{2};
        if constexpr (GUESS_HANDLES)
            return surf2Dread<uchar4>(imageID+1, coords.x<<MULT_FOUR_SHIFT, coords.y, hipBoundaryModeClamp);
        else   
            return surf2Dread<uchar4>(inputSurfaces[imageID], coords.x<<MULT_FOUR_SHIFT, coords.y, hipBoundaryModeClamp);
    }
    
    __device__ unsigned char loadPxFromMap(int mapID, int2 coords)
    {
        constexpr int MULT_FOUR_SHIFT{2};
        return surf2Dread<uchar4>(mapSurfaces[mapID], coords.x<<MULT_FOUR_SHIFT, coords.y, hipBoundaryModeClamp).x;
    }
   
    /* 
    __constant__ hipTextureObject_t inputTextures[MAX_SURFACES];
    template <typename T>
    __device__ PixelArray<T> loadPx(int imageID, int2 inCoords)
    {
        float2 coords{static_cast<float>(inCoords.x), static_cast<float>(inCoords.y)};
        if constexpr (GUESS_HANDLES)
            return PixelArray<T>{tex2D<uchar4>(imageID+1, coords.x+0.5f, coords.y+0.5f)};
        else    
            return PixelArray<T>{tex2D<uchar4>(inputTextures[imageID], coords.x, coords.y)};
    }
    __device__ uchar4 loadPx(int imageID, int2 inCoords)
    {
        float2 coords{static_cast<float>(inCoords.x), static_cast<float>(inCoords.y)};
        if constexpr (GUESS_HANDLES)
            return tex2D<uchar4>(imageID+1, coords.x+0.5f, coords.y+0.5f);
        else   
            return tex2D<uchar4>(inputTextures[imageID], coords.x, coords.y);
    }
   */ 

    __device__ void storePx(uchar4 px, int imageID, int2 coords)
    {
        if constexpr (GUESS_HANDLES)
            surf2Dwrite<uchar4>(px, imageID+1+gridSize(), coords.x*sizeof(uchar4), coords.y);
        else    
            surf2Dwrite<uchar4>(px, outputSurfaces[imageID], coords.x*sizeof(uchar4), coords.y);
    }
    
    __device__ void storePxToMap(uchar4 px, int mapID, int2 coords)
    {
            surf2Dwrite<uchar4>(px, mapSurfaces[mapID], coords.x*sizeof(uchar4), coords.y);
    }

    __device__ float distance(PixelArray<float> &a, PixelArray<float> &b)
    {
        return fmaxf(fmaxf(fabsf(a[0]-b[0]), fabsf(a[1]-b[1])), fabsf(a[2]-b[2]));
    }
/*
    template<typename T>
 class ElementRange 
        {
            private:
            float n{0};
            PixelArray<T> m;
            float m2{0};
            
            public:
            __device__ void add(PixelArray<T> val)
            {
               float dist = distance(m, val);
               n++;
               PixelArray delta = val-m;
               m += delta/static_cast<float>(n);
               //m2 += distance * Pixel::distance(m, val);
               m2 = __fmaf_rn(dist, distance(m, val), m2);

            }
            __device__ float dispersionAmount()
            {
                return m2/(n-1);    
            }      
            __device__ ElementRange& operator+=(const PixelArray<T>& rhs){

              add(rhs);
              return *this;
            }
        };
*/
    template<typename T>
    class ElementRange
    {
        private:
        PixelArray<T> minCol{float4{FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX}};
        PixelArray<T> maxCol{float4{FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN}};
        
        public:
        __device__ void add(PixelArray<T> val)
        {
            minCol[0] = fminf(minCol[0],val[0]);
            minCol[1] = fminf(minCol[1],val[1]);
            minCol[2] = fminf(minCol[2],val[2]);
            maxCol[0] = fmaxf(maxCol[0],val[0]);
            maxCol[1] = fmaxf(maxCol[1],val[1]);
            maxCol[2] = fmaxf(maxCol[2],val[2]);
        }
        __device__ float dispersionAmount()
        {
            return distance(minCol, maxCol); 
        }      
        __device__ ElementRange& operator+=(const PixelArray<T>& rhs){

          add(rhs);
          return *this;
        }
    };

    __device__ float focusDispersion(float focus, int2 coords)
    {
        int2 radius = blockRadius();
        constexpr int BLOCK_DIAMETER{3};
        constexpr int BLOCK_SIZE{BLOCK_DIAMETER*BLOCK_DIAMETER};
        ElementRange<float> dispersions[BLOCK_SIZE];

        for(int viewID= 0; viewID<FOCUS_MAP_IDS_COUNT; viewID++)
        {
            int gridID{focusMapIDs[viewID]};
            int i{0};
            int2 focusedCoords = focusCoords(coords, gridID, focus);
            for(int x = focusedCoords.x-radius.x; x <= focusedCoords.x+radius.x; x+=radius.x) 
                for(int y = focusedCoords.y-radius.y; y <= focusedCoords.y+radius.y; y+=radius.y)
                   dispersions[i++].add(loadPx<float>(gridID, {x,y}));
        }

        float finalDispersion{0};
        for(int i=0; i<BLOCK_SIZE; i++)
            finalDispersion += dispersions[i].dispersionAmount();
        return finalDispersion;
    }

    class MinDispersion
    {
        private:
        float dispersion{FLT_MAX};
        int focus{5};
        public:
        __device__ void add(int newFocus, float newDispersion)
        {
           if(newDispersion < dispersion)
           {
                focus = newFocus;
                dispersion = newDispersion;
           }
        }
        __device__ int getBestFocus()
        {
            return focus;
        } 
    };

    __global__ void estimateFocusMap()
    {
        int2 coords = getImgCoords();
        if(coordsOutside(coords))
            return;

        int step = focusRange()/32;
        MinDispersion minimum;
        for(int f=focus(); f<focus()+focusRange(); f+=step)
           minimum.add(f, focusDispersion(f, coords));

        int bestFocus = minimum.getBestFocus();
        float normalizedFocus = (bestFocus-focus())/static_cast<float>(focusRange());
        unsigned char mapFocus{static_cast<unsigned char>(round(normalizedFocus*UCHAR_MAX))};
        storePxToMap({mapFocus, mapFocus, mapFocus, UCHAR_MAX}, 0, coords);
        //loadPxFromMap(0, coords); 
    }

    template<bool allFocus>
    __global__ void process(half *weights)
    {
        int2 coords = getImgCoords();
        if(coordsOutside(coords))
            return;

        MemoryPartitioner<half> memoryPartitioner(localMemory);
        auto localWeights = memoryPartitioner.array(weightsSize());
        loadWeightsSync<half>(weights, localWeights);  
        PixelArray<float> sum[VIEW_TOTAL_COUNT];
        
        int2 focusedCoords;
        int focusValue;
        if constexpr (allFocus)
            focusValue = round((static_cast<float>(loadPxFromMap(0, coords))/UCHAR_MAX)*focusRange());

        for(int gridID = 0; gridID<gridSize(); gridID++)
        { 
            if constexpr (allFocus)
                focusedCoords = focusCoords(coords, gridID, focusValue);
            else
                focusedCoords = focusCoords(coords, gridID);

            auto px{loadPx<float>(gridID, focusedCoords)};
            for(int viewID=0; viewID<VIEW_TOTAL_COUNT; viewID++)
                    sum[viewID].addWeighted(localWeights[linearCoords({gridID,viewID}, weightsRes().x)], px);
        }

        for(int viewID=0; viewID<VIEW_TOTAL_COUNT; viewID++)
            storePx(sum[viewID].uch4(), viewID, coords);
    }

    __device__ half clamp(half value, float minimum, float maximum)
    {
        return max(min(value, maximum), minimum);
    }


    namespace Tensors
    {  
 
    constexpr int CHANNELS{3};
    constexpr int WARP_WIDTH{32};
    constexpr int WARP_COUNT = 256/WARP_WIDTH;
    constexpr int PIXELS{32}, VIEWS{8}, IMAGES{16};
    constexpr int PIXEL_MATRIX_SIZE{PIXELS*IMAGES};

    template<bool allFocus> 
    __device__ void loadPixels(int batch, int2 coords, unsigned char *destinationPixels, int focusValue)
    {
        int2 focusedCoords;
        const int batchOffset{batch*IMAGES};
        for(int image=0; image<IMAGES; image++) 
        {
            const int gridID{batchOffset+image};
            if constexpr (allFocus)
                focusedCoords = focusCoords(coords, gridID, focusValue);
            else
                focusedCoords = focusCoords(coords, gridID);

            uchar4 px = loadPx(gridID, focusedCoords);
            for(int channel=0; channel<CHANNELS; channel++)
                destinationPixels[IMAGES*channel + image] = reinterpret_cast<unsigned char*>(&px)[channel];
        }
    }

    __device__ void pixelsToSharedMemory(int channel, unsigned char *sourcePixels, int warpThreadID, half *currentLocalPixelsMemory, int pixelRowIDInt4)
    {
        const int linear = channel*IMAGES;
        int4 packed[2];
        for(int j=0; j<8; j++)
        {
            int jj = j<<1;
            reinterpret_cast<half2*>(&packed)[j] = half2{sourcePixels[linear+jj], sourcePixels[linear+jj+1]};
        }
        int bankA = warpThreadID%2;
        int bankB = (warpThreadID+1)%2;
        reinterpret_cast<int4*>(currentLocalPixelsMemory)[pixelRowIDInt4+bankA] = packed[bankA];
        reinterpret_cast<int4*>(currentLocalPixelsMemory)[pixelRowIDInt4+bankB] = packed[bankB];
    }

    __device__ void storePortionViews(int portion, int2 coords, half *pixels)
    {
        for(int viewID = 0; viewID<VIEW_COUNT; viewID++)
        {
            uchar4 color{0,0,0,255};
            for(int channel=0; channel<CHANNELS; channel++)
                reinterpret_cast<unsigned char*>(&color)[channel] = reinterpret_cast<half*>(pixels)[VIEWS*channel+viewID];
            storePx(color, viewID+portion*VIEW_COUNT, coords);
        }
    }

    template<bool allFocus>
    __global__ void process(half *weights)
    {
        using namespace nvcuda;

        int2 coords = getImgCoords();
        if(coordsOutside(coords))
            return;

        const int linearCoords = threadIdx.x+threadIdx.y*blockDim.x;
        const int warpID = linearCoords/WARP_WIDTH;
        const int warpThreadID = linearCoords%WARP_WIDTH;
        //const int warpCount = blockDim.x*blockDim.y/WARP_WIDTH;
        
        MemoryPartitioner<half> memoryPartitioner(localMemory);
        auto localPixelsMemory = memoryPartitioner.array(PIXEL_MATRIX_SIZE*WARP_COUNT);
        const int pixelRowIDInt4{(IMAGES>>3)*warpThreadID};
        half *currentLocalPixelsMemory = localPixelsMemory+(warpID*PIXEL_MATRIX_SIZE);;
        auto localWeights = memoryPartitioner.array(weightsSize());
        loadWeightsSync<half>(weights, localWeights); 
        
        //ROWSxCOLS
        //PIXELSxIMAGES
        wmma::fragment<wmma::matrix_a, PIXELS, VIEWS, IMAGES, half, wmma::row_major> matPixels;
        //IMAGES(weights)xVIEWS
        wmma::fragment<wmma::matrix_b, PIXELS, VIEWS, IMAGES, half, wmma::col_major> matWeights[VIEW_PORTIONS];
        //PIXELSxVIEWS
        wmma::fragment<wmma::accumulator, PIXELS, VIEWS, IMAGES, half> matResult[CHANNELS*VIEW_PORTIONS];
        for(int portion=0; portion<VIEW_PORTIONS; portion++)
            for(int channel=0; channel<CHANNELS; channel++) 
                wmma::fill_fragment(matResult[channel+portion*CHANNELS], 0.0f);
    
        uchar4 pixels[IMAGES]; 
        int focusValue; 
        if constexpr (allFocus)
            focusValue = round((static_cast<float>(loadPxFromMap(0, coords))/UCHAR_MAX)*focusRange());

        const int batchCount{gridSize()>>4}; // division by IMAGES
        for(int batch=0; batch<batchCount; batch++)
        {
            loadPixels<allFocus>(batch, coords, reinterpret_cast<unsigned char*>(&pixels), focusValue);
            for(int portion=0; portion<VIEW_PORTIONS; portion++)
                wmma::load_matrix_sync(matWeights[portion], localWeights+batch*IMAGES+portion*gridSize()*VIEWS, gridSize());

            for(int channel=0; channel<CHANNELS; channel++) 
            {
                pixelsToSharedMemory(channel, reinterpret_cast<unsigned char*>(pixels), warpThreadID, currentLocalPixelsMemory, pixelRowIDInt4);
                wmma::load_matrix_sync(matPixels, currentLocalPixelsMemory, IMAGES);
                for(int portion=0; portion<VIEW_PORTIONS; portion++)
                {
                    const int resultID = channel+portion*CHANNELS;
                    wmma::mma_sync(matResult[resultID], matPixels, matWeights[portion], matResult[resultID]);
                }
            }
        }

        for(int portion=0; portion<VIEW_PORTIONS; portion++)
        {
            for(int channel=0; channel<CHANNELS; channel++) 
            {
                wmma::store_matrix_sync(currentLocalPixelsMemory, matResult[channel+portion*CHANNELS], VIEWS, wmma::mem_row_major);
                reinterpret_cast<int4*>(pixels)[channel] = reinterpret_cast<int4*>(currentLocalPixelsMemory)[warpThreadID];
            }
            storePortionViews(portion, coords, reinterpret_cast<half*>(&pixels));
        }
 
    }
    }
}
