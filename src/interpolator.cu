#define GLM_FORCE_SWIZZLE
#include <sstream>
#include <hip/hip_runtime.h>
#include "lfLoader.h"
#include "interpolator.h"
#include "kernels.cu"
#include "libs/loadingBar/loadingbar.hpp"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb_image_write.h"

class Timer
{
    public:
    Timer()
    {    
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent);
    }
    float stop()
    {
        hipEventRecord(stopEvent);
        hipEventSynchronize(stopEvent);
        float time = 0;
        hipEventElapsedTime(&time, startEvent, stopEvent);
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);
        return time; 
    };
    private:
    hipEvent_t startEvent, stopEvent;
};

Interpolator::Interpolator(std::string inputPath) : input{inputPath}
{
    init();
}

Interpolator::~Interpolator()
{
    hipDeviceReset();
}

void Interpolator::init()
{
    loadGPUData();
    loadGPUConstants();
}

int Interpolator::createTextureObject(const uint8_t *data, glm::ivec3 size)
{
    hipChannelFormatDesc channels = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray *arr;
    hipMallocArray(&arr, &channels, size.x, size.y);
    hipMemcpy2DToArray(arr, 0, 0, data, size.x*size.z, size.x*size.z, size.y, hipMemcpyHostToDevice);
    
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = arr;
    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;
    hipTextureObject_t texObj{0};
    hipCreateTextureObject(&texObj, &texRes, &texDescr, NULL);
    return texObj;
}

std::pair<int, int*> Interpolator::createSurfaceObject(glm::ivec3 size)
{
    hipChannelFormatDesc channels = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned); 
    hipArray *arr;
    hipMallocArray(&arr, &channels, size.x, size.y, hipArraySurfaceLoadStore);

    hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(hipResourceDesc));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = arr;
    hipSurfaceObject_t surfObj = 0;
    hipCreateSurfaceObject(&surfObj, &surfRes);
    return {surfObj, reinterpret_cast<int*>(arr)};
}

void Interpolator::loadGPUData()
{
    LfLoader lfLoader;
    lfLoader.loadData(input);
    colsRows = lfLoader.getColsRows();
    resolution = lfLoader.imageResolution();

    std::cout << "Uploading data to GPU..." << std::endl;
    LoadingBar bar(lfLoader.imageCount()+viewCount);

    std::vector<hipSurfaceObject_t> surfaces;
    for(size_t i=0; i<viewCount; i++)
    {
        auto surface = createSurfaceObject(resolution);
        surfaces.push_back(surface.first);  
        outputArrays.push_back(surface.second);
        bar.add();
    }
    hipMalloc(&surfaceObjectsArr, surfaces.size()*sizeof(hipTextureObject_t));
    hipMemcpy(surfaceObjectsArr, surfaces.data(), surfaces.size()*sizeof(hipSurfaceObject_t), hipMemcpyHostToDevice);

    std::vector<hipTextureObject_t> textures;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        { 
            textures.push_back(createTextureObject(lfLoader.image({col, row}).data(), resolution)); 
            bar.add();
        }
    hipMalloc(&textureObjectsArr, textures.size()*sizeof(hipTextureObject_t));
    hipMemcpy(textureObjectsArr, textures.data(), textures.size()*sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
}

void Interpolator::loadGPUConstants()
{
    std::vector<int> values{resolution.x, resolution.y, colsRows.x, colsRows.y};
    hipMemcpyToSymbol(HIP_SYMBOL(Kernels::constants), values.data(), values.size() * sizeof(int));
}

std::vector<float> Interpolator::generateWeights(glm::vec2 coords)
{
    auto maxDistance = glm::distance(glm::vec2(0,0), glm::vec2(colsRows));
    float weightSum{0};
    std::vector<float> weightVals;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        {
            float weight = maxDistance - glm::distance(coords, glm::vec2(col, row));
            weightSum += weight;
            weightVals.push_back(weight);
        }
    for(auto &weight : weightVals)
        weight /= weightSum;
    return weightVals;
}

std::vector<glm::vec2> Interpolator::generateTrajectory(glm::vec4 startEndPoints)
{
    glm::vec2 step = (startEndPoints.zw() - startEndPoints.xy())/static_cast<float>(viewCount);
    std::vector<glm::vec2> trajectory;
    for(size_t i=0; i<viewCount; i++)
        trajectory.push_back(startEndPoints.xy()+step*static_cast<float>(i));
    return trajectory;
}

void Interpolator::loadGPUWeights(glm::vec4 startEndPoints)
{
    hipMalloc(reinterpret_cast<void **>(&weights), sizeof(half)*viewCount*colsRows.x*colsRows.y);
    auto trajectory = generateTrajectory(startEndPoints);
    std::vector<half> weightsMatrix;
    for(auto const &coord : trajectory)
    {
        auto floatWeightsLine = generateWeights(coord);
        std::vector<half> weightsLine;
        for(const auto & w : floatWeightsLine)
            weightsLine.push_back(static_cast<half>(w));
        weightsMatrix.insert(weightsMatrix.end(), weightsLine.begin(), weightsLine.end());
    }
    hipMemcpy(weights, weightsMatrix.data(), weightsMatrix.size(), hipMemcpyHostToDevice);
}

void Interpolator::interpolate(std::string outputPath, std::string trajectory, bool tensor)
{
    auto trajectoryPoints = interpretTrajectory(trajectory);
    loadGPUWeights(trajectoryPoints);
    
    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(resolution.x/dimBlock.x, resolution.y/dimBlock.y, 1);

    Timer timer;
    Kernels::process<<<dimGrid, dimBlock, 0>>>(reinterpret_cast<hipTextureObject_t*>(textureObjectsArr), reinterpret_cast<hipSurfaceObject_t*>(surfaceObjectsArr), reinterpret_cast<half*>(weights));
    std::cout << "Elapsed time: " << timer.stop() << " ms" << std::endl;
    storeResults(outputPath);
}

void Interpolator::storeResults(std::string path)
{
    std::cout << "Storing results..." << std::endl;
    LoadingBar bar(viewCount);
    std::vector<uint8_t> data(resolution.x*resolution.y*resolution.z, 255);
    for(size_t i=0; i<viewCount; i++) 
    {
        hipMemcpy2DFromArray(data.data(), resolution.x*resolution.z, reinterpret_cast<hipArray*>(outputArrays[i]), 0, 0, resolution.x*resolution.z, resolution.y, hipMemcpyDeviceToHost);
        stbi_write_png((path+std::to_string(i)+".png").c_str(), resolution.x, resolution.y, resolution.z, data.data(), resolution.x*resolution.z);
        bar.add();
    }
}

glm::vec4 Interpolator::interpretTrajectory(std::string trajectory)
{
    constexpr char delim{','};
    std::vector <std::string> numbers;
    std::stringstream a(trajectory); 
    std::string b; 
    while(getline(a, b, delim))
    {
        numbers.push_back(b);
    }
    glm::vec4 absolute;    
    int i{0};
    for (const auto &number : numbers)
    {
        float value = std::stof(number);
        absolute[i] = value*colsRows[i%2];
        i++;
    }
    return absolute;
}

